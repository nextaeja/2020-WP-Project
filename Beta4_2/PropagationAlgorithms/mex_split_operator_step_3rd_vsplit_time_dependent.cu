#include "hip/hip_runtime.h"
#include <mex.h>
#include <matrix.h>
#include <math.h>
#include <hipfft/hipfft.h>

#include "../MEX_helpers/complex.h"
#include "../MEX_helpers/cuda_helper.h"
#include "../MEX_helpers/interpolation1d.h"
#include "../Setup/cuda_setup_dynamic_potential.h"

#define NDIMS 3


__global__ void compute_expv(myComplex *dev_expv, double scale, size_t size);

void split_operator_3rd_vsplit_time(myComplex *dev_psi, myComplex *dev_expv, myComplex *dev_expk, double *dev_gauss_time,
		double *dev_gauss_pos, double *dev_x0, double *dev_y0, double *dev_z_offset, double t_query, double A, double eV,
		double expv_scale, size_t size, hipfftHandle forward_plan, hipfftHandle inverse_plan, const mwSize *gauss_dims,
		int nx, int ny, int nz, int decay_type, double dx, double dy, double dz, double dt) {
	double alpha = 2.0;

	/// TODO: UpdateBrownianMotionGaussians
	interpolate1d_adsorbate_positions<<<1, gauss_dims[0]>>>(dev_gauss_time, dev_gauss_pos, gauss_dims[0], gauss_dims[2], dev_x0, dev_y0, t_query, ny);
	hipDeviceSynchronize();
	setup_dynamic_gaussian_potential(dev_expv, dev_z_offset, dev_x0, dev_y0, gauss_dims[0], nx, ny, nz, decay_type, alpha, eV, A, dx, dy, dz);
	hipDeviceSynchronize();

	// Get the exponential of the potential
	compute_expv<<<NUM_BLOCKS, NUM_THREADS>>>(dev_expv, expv_scale, size);
	hipDeviceSynchronize();

	// Apply half potential operator
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expv, size);
	hipDeviceSynchronize();

	// Compute the forward FFT
	hipfftExecZ2Z(forward_plan, dev_psi, dev_psi, HIPFFT_FORWARD);
	hipDeviceSynchronize();

	// apply kinetic operator
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expk, size);
	hipDeviceSynchronize();

	// Invert FFT
	hipfftExecZ2Z(inverse_plan, dev_psi, dev_psi, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	complex_scale<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, 1/(double) size, size);
	hipDeviceSynchronize();

	/// TODO: UpdateBrownianMotionGaussians
	interpolate1d_adsorbate_positions<<<1, gauss_dims[0]>>>(dev_gauss_time, dev_gauss_pos, gauss_dims[0], gauss_dims[2], dev_x0, dev_y0, t_query+dt, ny);
	hipDeviceSynchronize();
	setup_dynamic_gaussian_potential(dev_expv, dev_z_offset, dev_x0, dev_y0, gauss_dims[0], nx, ny, nz, decay_type, alpha, eV, A, dx, dy, dz);
	hipDeviceSynchronize();

	// Get the exponential of the potential
	compute_expv<<<NUM_BLOCKS, NUM_THREADS>>>(dev_expv, expv_scale, size);
	hipDeviceSynchronize();

	// Apply half potential operator
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expv, size);
	hipDeviceSynchronize();
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	// Parse input parameters
	double t_query = mxGetScalar(prhs[0]);
	long long expv_ptr = mxGetScalar(prhs[1]);
	long long z_offset_ptr = mxGetScalar(prhs[2]);
	long long x0_ptr = mxGetScalar(prhs[3]);
	long long y0_ptr = mxGetScalar(prhs[4]);
	long long expk_ptr = mxGetScalar(prhs[5]);
	long long psi_ptr = mxGetScalar(prhs[6]);
	int nx = mxGetScalar(prhs[7]);
	int ny = mxGetScalar(prhs[8]);
	int nz = mxGetScalar(prhs[9]);
	int decay_type = mxGetScalar(prhs[10]);
	double A = mxGetScalar(prhs[11]);
	double eV = mxGetScalar(prhs[12]);
	double h_bar = mxGetScalar(prhs[13]);
	double dt = mxGetScalar(prhs[14]);
	double *gaussian_times = mxGetPr(prhs[15]);
	double *gaussian_positions = mxGetPr(prhs[16]);
	double dx = mxGetScalar(prhs[17]);
	double dy = mxGetScalar(prhs[18]);
	double dz = mxGetScalar(prhs[19]);

	double expv_scale = -dt / (2 * h_bar);

	// Calculate grid size
	size_t grid_size = nx * ny * nz;

	// Get number adsorbates
	const mwSize *gauss_dims = mxGetDimensions(prhs[16]);

	// Parse the pointers
	myComplex *dev_expv = reinterpret_cast<myComplex *>(expv_ptr);
	double *dev_z_offset = reinterpret_cast<double *>(z_offset_ptr);
	double *dev_x0 = reinterpret_cast<double *>(x0_ptr);
	double *dev_y0 = reinterpret_cast<double *>(y0_ptr);
	myComplex *dev_expk = reinterpret_cast<myComplex *>(expk_ptr);
	myComplex *dev_psi = reinterpret_cast<myComplex *>(psi_ptr);

	// Copy the adsorbate position and times into GPU
	// TODO: move this out of this function. Only execute at beginning
	double *dev_gauss_time, *dev_gauss_pos;
	hipMallocManaged(reinterpret_cast<void **>(&dev_gauss_time), gauss_dims[0] * gauss_dims[1] * gauss_dims[2] * sizeof(double));
	hipMallocManaged(reinterpret_cast<void **>(&dev_gauss_pos), gauss_dims[0] * gauss_dims[1] * gauss_dims[2] * sizeof(double));
	hipMemcpy(dev_gauss_time, gaussian_times, gauss_dims[0] * gauss_dims[1] * gauss_dims[2] * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_gauss_pos, gaussian_positions, gauss_dims[0] * gauss_dims[1] * gauss_dims[2] * sizeof(double), hipMemcpyHostToDevice);
	if (hipGetLastError() != hipSuccess) {
		mexErrMsgIdAndTxt("", "Memory allocation failed");
	}

	// Plan the FFT
	hipfftHandle forward_plan, inverse_plan;
	int n[3] = {nz, ny, nx};
	int idist = grid_size;
	int odist = grid_size;
	int istride = 1;
	int ostride = 1;
	int inembed[3] = {nz, ny, nx}; // MATLAB inverts rows and columns
	int onembed[3] = {nz, ny, nx};
	if (hipfftPlanMany(&forward_plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, 1) != HIPFFT_SUCCESS) {
		mexErrMsgIdAndTxt("", "Forward plan failed");
	}
	if (hipfftPlanMany(&inverse_plan, 3, n, onembed, ostride, odist, inembed, istride, idist, HIPFFT_Z2Z, 1) != HIPFFT_SUCCESS) {
		mexErrMsgIdAndTxt("", "Forward plan failed");
	}

	// TODO: PERFORM ACTUAL STEP
	split_operator_3rd_vsplit_time(dev_psi, dev_expv, dev_expk, dev_gauss_time, dev_gauss_pos, dev_x0, dev_y0, dev_z_offset, t_query, A, eV, expv_scale, grid_size, forward_plan, inverse_plan, gauss_dims, nx, ny, nz, decay_type, dx, dy, dz, dt);

	hipfftDestroy(forward_plan);
	hipfftDestroy(inverse_plan);
	hipFree(dev_gauss_time);
	hipFree(dev_gauss_pos);
}

__global__ void compute_expv(myComplex *dev_expv, double scale, size_t size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	while (tid < size) {
		myComplex expv;
		expv.x = cos(scale * dev_expv[tid].x);
		expv.y = sin(scale * dev_expv[tid].x);

		dev_expv[tid] = expv;

		tid += blockDim.x * gridDim.x;
	}
}
