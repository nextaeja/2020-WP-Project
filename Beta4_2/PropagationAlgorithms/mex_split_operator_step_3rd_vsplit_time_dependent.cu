#include "hip/hip_runtime.h"
#include <mex.h>
#include <matrix.h>
#include <math.h>
#include <hipfft/hipfft.h>

#include "../MEX_helpers/complex.h"
#include "../MEX_helpers/cuda_helper.h"

#define NDIMS 3


__global__ void compute_expv(myComplex *dev_expv, double scale, size_t size);


void split_operator_3rd_vsplit_time(myComplex *dev_psi, myComplex *dev_expv, myComplex *dev_expk, double A, double eV,
		double expv_scale, size_t size, hipfftHandle forward_plan, hipfftHandle inverse_plan) {
	double alpha = 2.0;
	double x_sigma = 3*(5.50/6)*A;
	double x_sigma = 3*(5.50/6)*A;
	double gauss_peak_val = 3*1.61*A;
	double well_depth = 10e-3*eV;

	/// TODO: UpdateBrownianMotionGaussians

	// Get the exponential of the potential
	compute_expv<<<NUM_BLOCKS, NUM_THREADS>>>(dev_expv, expv_scale, size);

	// Apply half potential operator
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expv, size);
	hipDeviceSynchronize();

	// Compute the forward FFT
	hipfftExecZ2Z(forward_plan, dev_psi, dev_psi, HIPFFT_FORWARD);

	// apply kinetic operator
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expk, size);
	hipDeviceSynchronize();

	// Invert FFT
	hipfftExecZ2Z(inverse_plan, dev_psi, dev_psi, HIPFFT_BACKWARD);
	complex_scale<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, 1/(double) size, size);

	/// TODO: UpdateBrownianMotionGaussians

	// Get the exponential of the potential
	compute_expv<<<NUM_BLOCKS, NUM_THREADS>>>(dev_expv, expv_scale, size);

	// Apply half potential operator
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expv, size);
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	// Parse input parameters
	double t = mxGetScalar(prhs[0]);
	long long expv_ptr = mxGetScalar(prhs[1]);
	long long z_offset_ptr = mxGetScalar(prhs[2]);
	long long x0_ptr = mxGetScalar(prhs[3]);
	long long y0_ptr = mxGetScalar(prhs[4]);
	long long expk_ptr = mxGetScalar(prhs[5]);
	long long psi_ptr = mxGetScalar(prhs[6]);
	size_t nx = mxGetScalar(prhs[7]);
	size_t ny = mxGetScalar(prhs[8]);
	size_t nz = mxGetScalar(prhs[9]);
	int decay_type = mxGetScalar(prhs[10]);
	double A = mxGetScalar(prhs[11]);
	double eV = mxGetScalar(prhs[12]);
	double h_bar;
	double dt;

	double expv_scale = -dt / (2 * h_bar);

	// Calculate grid size
	size_t grid_size = nx * ny * nz;

	// Plan the FFT
	hipfftHandle forward_plan, inverse_plan;
	int n[3] = {nz, ny, nx};
	int idist = size;
	int odist = size;
	int istride = 1;
	int ostride = 1;
	int inembed[3] = {nz, ny, nx}; // MATLAB inverts rows and columns
	int onembed[3] = {nz, ny, nx};
	hipfftPlanMany(&forward_plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, 1);
	hipfftPlanMany(&inverse_plan, 3, n, onembed, ostride, odist, inembed, istride, idist, HIPFFT_Z2Z, 1);

	// TODO: PERFORM ACTUAL STEP

	hipfftDestroy(forward_plan);
	hipfftDestroy(inverse_plan);
}

void update_brownian_motion_gaussians(myComplex *dev_v, size_t size, double *dev_x0, double *dev_y0, double t_query, int n_adsorbates) {
}

__global__ void compute_expv(myComplex *dev_expv, double scale, size_t size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	while (tid < size) {
		myComplex expv;
		expv.x = cos(scale * dev_expv[tid].x);
		expv.y = sin(scale * dev_expv[tid].x);

		dev_expv[tid] = expv;

		tid += blockDim.x * gridDim.x;
	}
}
