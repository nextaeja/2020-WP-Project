#include "hip/hip_runtime.h"
#include <mex.h>
#include <matrix.h>
#include <math.h>
#include <hipfft/hipfft.h>

#include "../MEX_helpers/complex.h"
#include "../MEX_helpers/cuda_helper.h"
#include "../Setup/cuda_setup_dynamic_potential.h"

const int NUM_GAUSSIAN_ADSORBATE_DIMENSIONS = 2;

__global__ void update_adsorbate_position(double *all_positions, double *dev_x0, double *dev_y0, int iteration, int num_adsorbates);
__device__ __host__ double _get_gaussian_adsorbate(double *data, int idx, int dim, int adsorbate, int num_adsorbates);
__global__ void compute_expv(myComplex *dev_expv, double scale, size_t size);

void split_operator_3rd_vsplit_time(myComplex *dev_psi, myComplex *dev_expv, myComplex *dev_expk,
		double *dev_x0, double *dev_y0, double *dev_z_offset, double t_query, double A, double eV,
		double expv_scale, size_t size, hipfftHandle forward_plan, hipfftHandle inverse_plan, int num_adsorbates,
		int nx, int ny, int nz, int decay_type, double dx, double dy, double dz, double dt) {
	double alpha = 2.0;

	setup_dynamic_gaussian_potential(dev_expv, dev_z_offset, dev_x0, dev_y0, num_adsorbates, nx, ny, nz, decay_type, alpha, eV, A, dx, dy, dz);

	// Get the exponential of the potential
	compute_expv<<<NUM_BLOCKS, NUM_THREADS>>>(dev_expv, expv_scale, size);

	// Apply half potential operator
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expv, size);

	// Compute the forward FFT
	CUDAFFT_HANDLE(hipfftExecZ2Z(forward_plan, dev_psi, dev_psi, HIPFFT_FORWARD));

	// apply kinetic operator
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expk, size);

	// Invert FFT
	CUDAFFT_HANDLE(hipfftExecZ2Z(inverse_plan, dev_psi, dev_psi, HIPFFT_BACKWARD));
	complex_scale<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, 1/(double) size, size);

	/// TODO: UpdateBrownianMotionGaussians
	setup_dynamic_gaussian_potential(dev_expv, dev_z_offset, dev_x0, dev_y0, num_adsorbates, nx, ny, nz, decay_type, alpha, eV, A, dx, dy, dz);

	// Get the exponential of the potential
	compute_expv<<<NUM_BLOCKS, NUM_THREADS>>>(dev_expv, expv_scale, size);

	// Apply half potential operator
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expv, size);
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	// Parse input parameters
	double t_query = mxGetScalar(prhs[0]);
	long long expv_ptr = mxGetScalar(prhs[1]);
	long long z_offset_ptr = mxGetScalar(prhs[2]);
	long long gauss_pos_ptr = mxGetScalar(prhs[3]);
	long long x0_ptr = mxGetScalar(prhs[4]);
	long long y0_ptr = mxGetScalar(prhs[5]);
	long long expk_ptr = mxGetScalar(prhs[6]);
	long long psi_ptr = mxGetScalar(prhs[7]);
	int nx = mxGetScalar(prhs[8]);
	int ny = mxGetScalar(prhs[9]);
	int nz = mxGetScalar(prhs[10]);
	int decay_type = mxGetScalar(prhs[11]);
	double A = mxGetScalar(prhs[12]);
	double eV = mxGetScalar(prhs[13]);
	double h_bar = mxGetScalar(prhs[14]);
	double dt = mxGetScalar(prhs[15]);
	double dx = mxGetScalar(prhs[16]);
	double dy = mxGetScalar(prhs[17]);
	double dz = mxGetScalar(prhs[18]);
	int iteration = mxGetScalar(prhs[19]);
	int num_adsorbates = mxGetScalar(prhs[20]);

	double expv_scale = -dt / (2 * h_bar);

	// Calculate grid size
	size_t grid_size = nx * ny * nz;

	// Parse the pointers
	myComplex *dev_expv = reinterpret_cast<myComplex *>(expv_ptr);
	double *dev_z_offset = reinterpret_cast<double *>(z_offset_ptr);
	double *dev_gauss_pos = reinterpret_cast<double *>(gauss_pos_ptr);
	double *dev_x0 = reinterpret_cast<double *>(x0_ptr);
	double *dev_y0 = reinterpret_cast<double *>(y0_ptr);
	myComplex *dev_expk = reinterpret_cast<myComplex *>(expk_ptr);
	myComplex *dev_psi = reinterpret_cast<myComplex *>(psi_ptr);

	// Plan the FFT
	hipfftHandle forward_plan, inverse_plan;
	int n[3] = {nz, ny, nx};
	int idist = grid_size;
	int odist = grid_size;
	int istride = 1;
	int ostride = 1;
	int inembed[3] = {nz, ny, nx}; // MATLAB inverts rows and columns
	int onembed[3] = {nz, ny, nx};
	CUDAFFT_HANDLE(hipfftPlanMany(&forward_plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, 1));
	CUDAFFT_HANDLE(hipfftPlanMany(&inverse_plan, 3, n, onembed, ostride, odist, inembed, istride, idist, HIPFFT_Z2Z, 1));

	// Compute the x and y positions of the adsorbates
	update_adsorbate_position<<<1, num_adsorbates>>>(dev_gauss_pos, dev_x0, dev_y0, iteration, num_adsorbates);

	split_operator_3rd_vsplit_time(dev_psi, dev_expv, dev_expk, dev_x0, dev_y0, dev_z_offset, t_query, A, eV, expv_scale, grid_size, forward_plan, inverse_plan, num_adsorbates, nx, ny, nz, decay_type, dx, dy, dz, dt);

	CUDAFFT_HANDLE(hipfftDestroy(forward_plan));
	CUDAFFT_HANDLE(hipfftDestroy(inverse_plan));
}

__global__ void update_adsorbate_position(double *all_positions, double *dev_x0, double *dev_y0, int iteration, int num_adsorbates) {
	int adsorbate = blockIdx.x * blockDim.x + threadIdx.x;

	while (adsorbate < num_adsorbates) {
		dev_x0[adsorbate] = _get_gaussian_adsorbate(all_positions, iteration, 0, adsorbate, num_adsorbates);
		dev_y0[adsorbate] = _get_gaussian_adsorbate(all_positions, iteration, 1, adsorbate, num_adsorbates);

		adsorbate += blockDim.x * gridDim.x;
	}
}

// The gaussian position array is a 3D one, return the correct value for a given
//	adsorbate number and dimension (x or y)
__device__ __host__ double _get_gaussian_adsorbate(double *data, int idx, int dim, int adsorbate, int num_adsorbates) {
	if (idx < 0) {
		return -1.0;
	}

	int tot_idx = adsorbate + dim*num_adsorbates + idx*num_adsorbates*NUM_GAUSSIAN_ADSORBATE_DIMENSIONS;

	return data[tot_idx];
}

__global__ void compute_expv(myComplex *dev_expv, double scale, size_t size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	while (tid < size) {
		myComplex expv;
		expv.x = cos(scale * dev_expv[tid].x);
		expv.y = sin(scale * dev_expv[tid].x);

		dev_expv[tid] = expv;

		tid += blockDim.x * gridDim.x;
	}
}
