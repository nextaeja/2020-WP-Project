#include <mex.h>
#include <matrix.h>
#include <math.h>
#include <hipfft/hipfft.h>

#include "../MEX_helpers/complex.h"
#include "../MEX_helpers/cuda_helper.h"

#define NDIMS 3

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	long long expv_ptr = mxGetScalar(prhs[0]);
	long long expk_ptr = mxGetScalar(prhs[1]);
	long long psi_ptr = mxGetScalar(prhs[2]);
	int nx = mxGetScalar(prhs[3]);
	int ny = mxGetScalar(prhs[4]);
	int nz = mxGetScalar(prhs[5]);
	size_t size = nx * ny * nz;

	// Parse the pointers
	hipComplex *dev_expv = reinterpret_cast<hipComplex *>(expv_ptr);
	hipComplex *dev_expk = reinterpret_cast<hipComplex *>(expk_ptr);
	hipComplex *dev_psi = reinterpret_cast<hipComplex *>(psi_ptr);

	// psiVStepHalf = expV.*psi;
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expv, size);
	hipDeviceSynchronize();

	// Compute the FFT
	hipfftHandle plan;
	int n[3] = {nx, ny, nz};
	hipfftPlan3d(&plan, nx, ny, nz, HIPFFT_C2C);
	/*
	hipfftPlanMany(&plan, 3, n,
					NULL, 1, size,
					NULL, 1, size,
					HIPFFT_C2C, 1);
	*/
	hipfftExecC2C(plan, dev_psi, dev_psi, HIPFFT_BACKWARD);

	/*
	// psiKStepFT = expK.*psiVStepHalfFT;
	complex_mul<<<NUM_BLOCKS, NUM_THREADS>>>(dev_psi, dev_expk, size);

	// Invert FFT
	hipfftExecC2C(plan, dev_psi, dev_psi, HIPFFT_BACKWARD);
	*/

	hipfftDestroy(plan);
}
