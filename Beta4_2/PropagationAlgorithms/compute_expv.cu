#include "hip/hip_runtime.h"
#include <mex.h>
#include <matrix.h>
#include <math.h>

#include "../MEX_helpers/complex.h"
#include "../MEX_helpers/cuda_helper.h"

// Compute exp((-1i*(dt/2)/hBar)*V)
__global__ void compute_expv(myComplex *potential, double scale, size_t size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	while (tid < size) {
		myComplex expv;
		expv.x = cos(scale * potential[tid].x);
		expv.y = sin(scale * potential[tid].x);

		potential[tid] = expv;

		tid += blockDim.x * gridDim.x;
	}
}

// expV = exp((-1i*(dt/2)/hBar)*V);
// Compute the exponential of the potential in place (no copy/allocation is needed)
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	// Exctract the parameters
	long long potential_ptr = mxGetScalar(prhs[0]);
	double h_bar = mxGetScalar(prhs[1]);
	double dt = mxGetScalar(prhs[2]);
	size_t size = mxGetScalar(prhs[3]);

	// Parse the pointer to allocated space for expK and k_squared
	myComplex *potential = reinterpret_cast<myComplex *>(potential_ptr);

	// Get scaling constant (-1i*(dt/2)/hBar)
	double scale = -(dt/2) / h_bar;

	// Calculate the exponential
	compute_expv<<<NUM_BLOCKS, NUM_THREADS>>>(potential, scale, size);
}
