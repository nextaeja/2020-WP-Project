#include "hip/hip_runtime.h"
#include <mex.h>
#include <matrix.h>
#include <math.h>

#include "../MEX_helpers/complex.h"
#include "../MEX_helpers/cuda_helper.h"

// Compute (1i*dt/hBar)*(hBar^2+kSquared/(2*mass))
__global__ void compute_expk(Complex *expk, double *ksquared, Complex prefactor, double h_bar, double mass, size_t size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	while (tid < size) {
		float shifted_k2 = h_bar*h_bar + ksquared[tid] / (2*mass);

		expk[tid] = _complex_mul(prefactor, shifted_k2);
		expk[tid] = _complex_exp(expk[tid]);

		tid += blockDim.x * gridDim.x;
	}
}

// Compute exp((-1i*dt/hBar)*(-hBar^2*-kSquared/(2*mass)))
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	// Exctract the parameters
	long long expk_ptr = mxGetScalar(prhs[0]);
	long long k_squared_ptr = mxGetScalar(prhs[1]);
	double h_bar = mxGetScalar(prhs[2]);
	double dt = mxGetScalar(prhs[3]);
	double mass = mxGetScalar(prhs[4]);
	double *k_squared = mxGetPr(prhs[5]);
	size_t size = mxGetScalar(prhs[6]);

	// Parse the pointer to allocated space for expK and k_squared
	Complex *dev_expk = reinterpret_cast<Complex *>(expk_ptr);
	double *dev_k_squared = reinterpret_cast<double *>(k_squared_ptr);

	// k_squared is computed in MATLAB and copied over the GPU
	hipMemcpy(dev_k_squared, k_squared, size * sizeof(double), hipMemcpyHostToDevice);

	// Get scaling constant (-1i*dt/hBar)
	Complex scale;
	scale.x = 0.0;
	scale.y = dt / h_bar;

	// Compute the argument of the exponential
	compute_expk<<<NUM_BLOCKS, NUM_THREADS>>>(dev_expk, dev_k_squared, scale, h_bar, mass, size);
}
