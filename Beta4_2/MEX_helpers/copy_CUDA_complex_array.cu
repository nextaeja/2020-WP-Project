#include "hip/hip_runtime.h"
/* Copyright (c) 2020, Lorenzo Basso, Jack Lee, Matthew Zhang, Feiyang Chen
 * Copyright (c) 2018, Francis Haghighi-Daly 
 * All rights reserved.
 * This file is part of the WooStOr - Wavepacket prOpopgatiOn using SpliT OperatR method, subject to the GNU/GPL-3.0-or-later.*/

#include <mex.h>
#include <matrix.h>
#include <math.h>

#include "../MEX_helpers/complex.h"
#include "../MEX_helpers/cuda_helper.h"

__global__ void copy_complex_array(myComplex *dest, double *real, double *imag, size_t size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	while (tid < size) {
		myComplex w;
		w.x = real[tid];
		w.y = imag[tid];
		dest[tid] = w;

		tid += blockDim.x * gridDim.x;
	}
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	long long dest_ptr = mxGetScalar(prhs[0]);
	double *source_real = mxGetPr(prhs[1]);
	double *source_imag = mxGetPi(prhs[1]);
	size_t size = mxGetScalar(prhs[2]);

	myComplex *dest = reinterpret_cast<myComplex *>(dest_ptr);

	// Allocate the space on the GPU
	double *dev_source_real, *dev_source_imag;
	hipMallocManaged(reinterpret_cast<void **>(&dev_source_real), size * sizeof(double));
	hipMallocManaged(reinterpret_cast<void **>(&dev_source_imag), size * sizeof(double));

	// Copy input data to GPU
	hipMemcpy(dev_source_real, source_real, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_source_imag, source_imag, size * sizeof(double), hipMemcpyHostToDevice);

	copy_complex_array<<<NUM_BLOCKS, NUM_THREADS>>>(dest, dev_source_real, dev_source_imag, size);

	hipFree(dev_source_real);
	hipFree(dev_source_imag);
}
