/* Copyright (c) 2020, Lorenzo Basso, Jack Lee, Matthew Zhang, Feiyang Chen
 * Copyright (c) 2018, Francis Haghighi-Daly 
 * All rights reserved.
 * This file is part of the WooStOr - Wavepacket prOpopgatiOn using SpliT OperatR method, subject to the GNU/GPL-3.0-or-later.*/

#include <mex.h>
#include <matrix.h>
#include <math.h>

#include "../MEX_helpers/complex.h"
#include "../MEX_helpers/cuda_helper.h"

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	long long potential_ptr = mxGetScalar(prhs[0]);
	int nx = mxGetScalar(prhs[1]);
	int ny = mxGetScalar(prhs[2]);
	int nz = mxGetScalar(prhs[3]);

	myComplex *dev_potential = reinterpret_cast<myComplex *>(potential_ptr);
	myComplex *potential = reinterpret_cast<myComplex *>(malloc(nx * ny * nz * sizeof(myComplex)));
	hipMemcpy(potential, dev_potential, nx * ny * nz * sizeof(myComplex), hipMemcpyDeviceToHost);

	for (int k=0; k<nz; k++) {
		for (int i=0; i<nx; i++) {
			for (int j=0; j<ny; j++) {
				int idx = k*nx*ny+j*nx+i;
				mexPrintf("(%e + i*%e) ", potential[idx].x, potential[idx].y);
			}
			mexPrintf("\n");
		}
		mexPrintf("\n");
	}

	free(potential);
}
