/* Copyright (c) 2020, Lorenzo Basso, Jack Lee, Matthew Zhang, Feiyang Chen
 * Copyright (c) 2018, Francis Haghighi-Daly 
 * All rights reserved.
 * This file is part of the WooStOr - Wavepacket prOpopgatiOn using SpliT OperatR method, subject to the GNU/GPL-3.0-or-later.*/

#include "cuda_helper.h"

void _handle_cuda_error(hipError_t code, const char *file, int line) {
	if (code != hipSuccess) {
		char err_msg[500];
		sprintf(err_msg, "Error '%s' occurred in file '%s'@%d\n", hipGetErrorString(code), file, line);
		mexErrMsgIdAndTxt("SplitOperator:CUDA:FFT", err_msg);
	}
}

void _handle_cudafft_error(hipfftResult code, const char *file, int line) {
	if (code != HIPFFT_SUCCESS) {
		char err_msg[500];
		sprintf(err_msg, "Cuda FFT error occurred in file '%s'@%d\n", file, line);
		mexErrMsgIdAndTxt("SplitOperator:CUDA:FFT", err_msg);
	}
}
