#include "cuda_helper.h"

void _handle_cuda_error(hipError_t code, const char *file, int line) {
	if (code != hipSuccess) {
		char err_msg[500];
		sprintf(err_msg, "Error '%s' occurred in file '%s'@%d\n", hipGetErrorString(code), file, line);
		mexErrMsgIdAndTxt("SplitOperator:CUDA:FFT", err_msg);
	}
}

void _handle_cudafft_error(hipfftResult code, const char *file, int line) {
	if (code != HIPFFT_SUCCESS) {
		char err_msg[500];
		sprintf(err_msg, "Cuda FFT error occurred in file '%s'@%d\n", file, line);
		mexErrMsgIdAndTxt("SplitOperator:CUDA:FFT", err_msg);
	}
}
