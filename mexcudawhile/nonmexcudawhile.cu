#include "hip/hip_runtime.h"
#include <mex.h>
#include <matrix.h>
#include <math.h>
#include <hipfft/hipfft.h>
#include "hip/hip_runtime.h"

#include "../MEX_helpers/complex.h"
#include "../MEX_helpers/cuda_helper.h"
#include "../Setup/cuda_setup_dynamic_potential.h"

//#include "unistd.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

const int NUM_GAUSSIAN_ADSORBATE_DIMENSIONS = 2;

__global__ void update_adsorbate_position(double* all_positions, double* dev_x0, double* dev_y0, int iteration, int num_adsorbates);
__device__ __host__ double _get_gaussian_adsorbate(double* data, int idx, int dim, int adsorbate, int num_adsorbates);
__global__ void compute_expv(myComplex* dev_expv, double scale, size_t size);

void split_operator_3rd_vsplit_time(myComplex* dev_psi, myComplex* dev_expv, myComplex* dev_expk,
	double* dev_x0, double* dev_y0, double* dev_z_offset, double t_query, double A, double eV,
	double expv_scale, size_t size, hipfftHandle forward_plan, hipfftHandle inverse_plan, const mwSize* gauss_dims,
	int nx, int ny, int nz, int decay_type, double dx, double dy, double dz, double dt,double alpha) {


	setup_dynamic_gaussian_potential(dev_expv, dev_z_offset, dev_x0, dev_y0, gauss_dims[0], nx, ny, nz, decay_type, alpha, eV, A, dx, dy, dz);

	// Get the exponential of the potential
	compute_expv << <NUM_BLOCKS, NUM_THREADS >> > (dev_expv, expv_scale, size);

	// Apply half potential operator
	complex_mul << <NUM_BLOCKS, NUM_THREADS >> > (dev_psi, dev_expv, size);

	// Compute the forward FFT
	CUDAFFT_HANDLE(hipfftExecZ2Z(forward_plan, dev_psi, dev_psi, HIPFFT_FORWARD));

	// apply kinetic operator
	complex_mul << <NUM_BLOCKS, NUM_THREADS >> > (dev_psi, dev_expk, size);

	// Invert FFT
	CUDAFFT_HANDLE(hipfftExecZ2Z(inverse_plan, dev_psi, dev_psi, HIPFFT_BACKWARD));
	complex_scale << <NUM_BLOCKS, NUM_THREADS >> > (dev_psi, 1 / (double)size, size);

	/// TODO: UpdateBrownianMotionGaussians
	setup_dynamic_gaussian_potential(dev_expv, dev_z_offset, dev_x0, dev_y0, gauss_dims[0], nx, ny, nz, decay_type, alpha, eV, A, dx, dy, dz);

	// Get the exponential of the potential
	compute_expv << <NUM_BLOCKS, NUM_THREADS >> > (dev_expv, expv_scale, size);

	// Apply half potential operator
	complex_mul << <NUM_BLOCKS, NUM_THREADS >> > (dev_psi, dev_expv, size);
}

void mex_split_operator_step_3rd_vsplit_time_dependent(double t_query, myComplex* dev_expv, double* dev_z_offset, double* dev_gauss_pos, double* dev_x0, double* dev_y0, myComplex* dev_expk, myComplex* dev_psi, int nx, int ny, int nz, int decay_type, double A, double eV, double h_bar, double dt, double dx, double dy, double dz, int iteration, const mwSize* gauss_dims,double alpha) {
	// Parse input parameters
	/*double t_query = mxGetScalar(prhs[0]);
	long long expv_ptr = mxGetScalar(prhs[1]);
	long long z_offset_ptr = mxGetScalar(prhs[2]);
	long long gauss_pos_ptr = mxGetScalar(prhs[3]);
	long long x0_ptr = mxGetScalar(prhs[4]);
	long long y0_ptr = mxGetScalar(prhs[5]);
	long long expk_ptr = mxGetScalar(prhs[6]);
	long long psi_ptr = mxGetScalar(prhs[7]);
	int nx = mxGetScalar(prhs[8]);
	int ny = mxGetScalar(prhs[9]);
	int nz = mxGetScalar(prhs[10]);
	int decay_type = mxGetScalar(prhs[11]);
	double A = mxGetScalar(prhs[12]);
	double eV = mxGetScalar(prhs[13]);
	double h_bar = mxGetScalar(prhs[14]);
	double dt = mxGetScalar(prhs[15]);
	double dx = mxGetScalar(prhs[16]);
	double dy = mxGetScalar(prhs[17]);
	double dz = mxGetScalar(prhs[18]);
	int iteration = mxGetScalar(prhs[19]);
	*/
	double expv_scale = -dt / (2 * h_bar);


	// Calculate grid size
	size_t grid_size = nx * ny * nz;

	// Get number adsorbates
	//const mwSize *gauss_dims = mxGetDimensions(prhs[15]);

	// Parse the pointers
	/*myComplex *dev_expv = reinterpret_cast<myComplex *>(expv_ptr);
	double *dev_z_offset = reinterpret_cast<double *>(z_offset_ptr);
	double *dev_gauss_pos = reinterpret_cast<double *>(gauss_pos_ptr);
	double *dev_x0 = reinterpret_cast<double *>(x0_ptr);
	double *dev_y0 = reinterpret_cast<double *>(y0_ptr);
	myComplex *dev_expk = reinterpret_cast<myComplex *>(expk_ptr);
	myComplex *dev_psi = reinterpret_cast<myComplex *>(psi_ptr);
	*/
	// Plan the FFT
	hipfftHandle forward_plan, inverse_plan;
	int n[3] = { nz, ny, nx };
	int idist = grid_size;
	int odist = grid_size;
	int istride = 1;
	int ostride = 1;
	int inembed[3] = { nz, ny, nx }; // MATLAB inverts rows and columns
	int onembed[3] = { nz, ny, nx };
	CUDAFFT_HANDLE(hipfftPlanMany(&forward_plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2Z, 1));
	CUDAFFT_HANDLE(hipfftPlanMany(&inverse_plan, 3, n, onembed, ostride, odist, inembed, istride, idist, HIPFFT_Z2Z, 1));

	// Compute the x and y positions of the adsorbates
	update_adsorbate_position << <1, gauss_dims[0] >> > (dev_gauss_pos, dev_x0, dev_y0, iteration, gauss_dims[0]);

	split_operator_3rd_vsplit_time(dev_psi, dev_expv, dev_expk, dev_x0, dev_y0, dev_z_offset, t_query, A, eV, expv_scale, grid_size, forward_plan, inverse_plan, gauss_dims, nx, ny, nz, decay_type, dx, dy, dz, dt,alpha);

	CUDAFFT_HANDLE(hipfftDestroy(forward_plan));
	CUDAFFT_HANDLE(hipfftDestroy(inverse_plan));
}

__global__ void update_adsorbate_position(double* all_positions, double* dev_x0, double* dev_y0, int iteration, int num_adsorbates) {
	int adsorbate = blockIdx.x * blockDim.x + threadIdx.x;

	while (adsorbate < num_adsorbates) {
		dev_x0[adsorbate] = _get_gaussian_adsorbate(all_positions, iteration, 0, adsorbate, num_adsorbates);
		dev_y0[adsorbate] = _get_gaussian_adsorbate(all_positions, iteration, 1, adsorbate, num_adsorbates);

		adsorbate += blockDim.x * gridDim.x;
	}
}

// The gaussian position array is a 3D one, return the correct value for a given
//	adsorbate number and dimension (x or y)
__device__ __host__ double _get_gaussian_adsorbate(double* data, int idx, int dim, int adsorbate, int num_adsorbates) {
	if (idx < 0) {
		return -1.0;
	}

	int tot_idx = adsorbate + dim * num_adsorbates + idx * num_adsorbates * NUM_GAUSSIAN_ADSORBATE_DIMENSIONS;

	return data[tot_idx];
}

__global__ void compute_expv(myComplex* dev_expv, double scale, size_t size) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	while (tid < size) {
		myComplex expv;
		expv.x = cos(scale * dev_expv[tid].x);
		expv.y = sin(scale * dev_expv[tid].x);

		dev_expv[tid] = expv;

		tid += blockDim.x * gridDim.x;
	}
}






void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]) {

	char cwdtmp[256];

	//getcwd(cwd, sizeof(cwd));
	char* cwd = "C:\\Users\\jackl\\Documents\\MATLAB\\summer 2020\\2020-WP-Project-6_optimizing_setup_dynamic_gaussian_potential\\Beta4_2\\mexcudawhile";//make general
	char* l = strrchr(cwd, '\\');
	cwd[strlen(cwd) - strlen(l) + 1] = '\0';

	strcpy(cwdtmp, cwd);

	FILE* setup = fopen(strcat(cwdtmp, "setup.txt"), "r");//navigates up to setup file


	char buff[1024];
	
	ps = e-12;

	fgets(buff, sizeof(buff), setup);
	expv_ptr = strtoll(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	z_offset_ptr = strtoll(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	gauss_pos_ptr = strtoll(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	x0_ptr = strtoll(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	y0_ptr = strtoll(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	expk_ptr = strtoll(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	psi_ptr = strtoll(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	nx = atoi(buff);

	fgets(buff, sizeof(buff), setup);
	ny = atoi(buff);

	fgets(buff, sizeof(buff), setup);
	nz = atoi(buff);
	
	fgets(buff, sizeof(buff), setup);
	decay_type = atoi(buff);

	fgets(buff, sizeof(buff), setup);
	A = strtod(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	eV = strtod(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	h_bar = strtod(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	dt = strtod(buff, NULL) * ps;

	fgets(buff, sizeof(buff), setup);
	dx = strtod(buff, NULL) * A;

	fgets(buff, sizeof(buff), setup);
	dy = strtod(buff, NULL) * A;

	fgets(buff, sizeof(buff), setup);
	dz = strtod(buff, NULL) * A;

	fgets(buff, sizeof(buff), setup);
	gfxSteps = strtol(buff, NULL);

	fgets(buff, sizeof(buff), setup);
	t = strtod(buff, NULL) * ps;

	fgets(buff, sizeof(buff), setup);
	alpha = strtod(buff, NULL);

	fclose(setup);



	int it = 0;
	//int* sum;

	myComplex* dev_expv = reinterpret_cast<myComplex*>(expv_ptr);
	double* dev_z_offset = reinterpret_cast<double*>(z_offset_ptr);
	double* dev_gauss_pos = reinterpret_cast<double*>(gauss_pos_ptr);
	double* dev_x0 = reinterpret_cast<double*>(x0_ptr);
	double* dev_y0 = reinterpret_cast<double*>(y0_ptr);
	myComplex* dev_expk = reinterpret_cast<myComplex*>(expk_ptr);
	myComplex* dev_psi = reinterpret_cast<myComplex*>(psi_ptr);

	//const mwSize* gauss_dims = mxGetDimensions(prhs[15]);

	const size_t gauss_dims = sizeof(dx);//no idea if this is actually a reasonable replacement

	while (it <= gfxSteps) {
		/*double totprob = 0;
		for (int x = 0;x < nx;x++) {
			for (int y = 0;y < ny;y++) {
				for (int z = 0;z < nz;z++) {
					for (int im = 0;im < 2;im++) {
						totprob = totprob + psi[x][y][z][im] * psi[x][y][z][im];
					}
				}
			}
		}


		if (!(fabs(totprob - 1) < eps)) {
			nrhs = 1;
			prhs[0] = 5;//make actual error
		}*/

		//* sum = 0;



		mex_split_operator_step_3rd_vsplit_time_dependent(t, dev_expv, dev_z_offset, dev_gauss_pos, dev_x0, dev_y0, dev_expk, dev_psi, nx, ny, nz, decay_type, A, eV, h_bar, dt, dx, dy, dz, it, gauss_dims,alpha);
		it++;
		t = t + dt;
	}

}